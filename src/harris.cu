#include "hip/hip_runtime.h"
#pragma once
#include "../include/harris.h"
#include "opencv2/opencv.hpp"
#include "../include/conv.h"
#include "../include/utils.h"
#include <math.h>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__));


// cornerHarris��������ÿһ�����أ�x,y)��blockSize x blockSize �����ڣ�
// ����2x2�ݶȵ�Э�������M(x,y)���Ϳ����ҳ����ͼ�еľֲ����ֵ�����ҳ��˽ǵ㡣

namespace harris 
{
	void opencv_harris(const char * address)
	{
		cv::Mat src_img = cv::imread(address);
		cv::cvtColor(src_img, src_img, cv::COLOR_BGR2GRAY);
		cv::Mat harris_img;
		cornerHarris(src_img, harris_img, 2, 3, 0.04, cv::BORDER_DEFAULT);
		// harris_img type = CV_32F
		printf("type = %d", harris_img.type());
		cv::Mat harris_bw_img;
		cv::threshold(harris_img, harris_bw_img, 0.00001, 255, cv::THRESH_BINARY);
		cv::namedWindow("bw", cv::WINDOW_NORMAL);
		cv::imshow("bw", harris_bw_img);

		cv::normalize(harris_img, harris_img, 0, 1, cv::NORM_MINMAX, CV_32FC1);

		cv::namedWindow("harris_img", cv::WINDOW_NORMAL);
		cv::imshow("harris_img", harris_img);
		cv::waitKey(0);
	}

	void cuda_harris(cv::Mat & src, cv::Mat & dst, const int & block_size, const float & prop, 
		const int &aperture_size)
	{
		
		hipSetDevice(0);
		// 3 x 3 Sobel operator
		//float * sobel_y = new float[aperture_size * aperture_size];
		float * sobel_y = (float *)malloc(aperture_size * aperture_size * sizeof(float));
		sobel_y[0] = -1.; sobel_y[1] = -2.; sobel_y[2] = -1.;
		sobel_y[3] =  0.; sobel_y[4] = 0.; sobel_y[5] = 0.;
		sobel_y[6] =  1.; sobel_y[7] = 2.; sobel_y[8] = 1.;
		//float * sobel_x = new float[aperture_size * aperture_size];
		float * sobel_x = (float *)malloc(aperture_size * aperture_size * sizeof(float));
		sobel_x[0] = -1.; sobel_x[1] = 0.; sobel_x[2] = 1.;
		sobel_x[3] = -2.; sobel_x[4] = 0.; sobel_x[5] = 2.;
		sobel_x[6] = -1.; sobel_x[7] = 0.; sobel_x[8] = 1.;
		
		// conv get gradient map.
		cv::Mat sobel_x_img;
		conv::cuda_conv(src, sobel_x_img, sobel_x, aperture_size);
		cv::Mat sobel_y_img;
		conv::cuda_conv(src, sobel_y_img, sobel_y, aperture_size);
		printf("conv get gradient map finish \n");
		// ///////////////////////////////////////////
		
		cv::namedWindow("sobel_x", cv::WINDOW_NORMAL);
		cv::imshow("sobel_x", sobel_x_img);
		cv::waitKey(0);



		int img_rows = src.rows;
		int img_cols = src.cols;
		printf("\n src_img size = [%d, %d]\n", img_rows, img_cols);
		printf("\n gradient_img size = [%d, %d]\n", sobel_x_img.rows, sobel_x_img.cols);


		size_t img_size = img_cols * img_rows * sizeof(float);
		float * sobel_x_img_vec;
		float * sobel_y_img_vec;
		float * gpu_result_vec;

		// memory allocate
		HANDLE_ERROR(hipMalloc((void **)& sobel_x_img_vec, img_size));
		HANDLE_ERROR(hipMalloc((void **)& sobel_y_img_vec, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result_vec, img_size));

		// memory copy 
		HANDLE_ERROR(hipMemcpy(sobel_x_img_vec, sobel_x_img.data, img_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(sobel_y_img_vec, sobel_y_img.data, img_size, hipMemcpyHostToDevice));


		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		printf("block_num = %d, thread_num = %d !\n", block_num, thread_num);
		dim3 thread_grid_size(block_num, 1, 1);
		dim3 thread_block_size(thread_num, 1, 1);
		
		harris::harris_kernal<<< thread_grid_size, thread_block_size >>>
			(sobel_x_img_vec, sobel_y_img_vec, gpu_result_vec, img_rows, img_cols, block_size, prop);
		
		
		printf("finish kernel!!!\n");
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
		{
			printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		// getLastCudaError("Kernel execution failed");

		float * cpu_result_vec = (float *)malloc(img_size);

		//HANDLE_ERROR(hipMemcpy(cpu_result_vec, gpu_result_vec, img_size, hipMemcpyDeviceToHost));
		hipMemcpy(cpu_result_vec, gpu_result_vec, img_size, hipMemcpyDeviceToHost);

		dst = cv::Mat(img_rows, img_cols, CV_32FC1, cpu_result_vec).clone();
		cv::normalize(dst, dst, 1.0, 0.0, cv::NORM_MINMAX);

		cv::namedWindow("dst_img", cv::WINDOW_NORMAL);
		cv::imshow("dst_img", dst);
		cv::waitKey(0);
		

		HANDLE_ERROR(hipFree(sobel_x_img_vec));
		HANDLE_ERROR(hipFree(sobel_y_img_vec));
		HANDLE_ERROR(hipFree(gpu_result_vec));
		free(cpu_result_vec);
		free(sobel_y);
		free(sobel_x);

		//delete[] cpu_result_vec;
		//delete[] sobel_y;
		//delete[] sobel_x;
		hipDeviceReset();
	}

	__global__ void harris_kernal(float * sobel_x_vec, float * sobel_y_vec, float * result_vec, 
		const int &img_row, const int &img_col, const int & block_size, 
		const float & prop) 
	{
		int thread_id = threadIdx.x;
		int block_id = blockIdx.x;
		int index = block_id * blockDim.x + thread_id;
		if (block_id == 0 ||thread_id == 0)
		{
			printf("thread_id = %d, block_id = %d, index = %d\n", thread_id, block_id, index);
		}
		if (index >= img_row * img_col)
		{
			return;
		}
		int pixel_col = index % img_col;
		int pixel_row = index / img_col;
		// calculate covariance matrix
		float gradient_x_sum = 0;
		float gradient_y_sum = 0;
		float gradient_xy_sum = 0;
		float gradient_xx_sum = 0;
		float gradient_yy_sum = 0;
		for (int i = 0; i < block_size; ++i)
		{
			for (int j = 0; j < block_size; ++j) 
			{
				int cur_row = pixel_row - block_size / 2 + i;
				int cur_col = pixel_col - block_size / 2 + j;
				//float 
				float gradient_x = 0.;
				float gradient_y = 0.;
				if (cur_row < 0 || cur_row >= img_row || cur_col < 0 || cur_col >= img_col) 
				{
				}
				else {
					gradient_x = sobel_x_vec[cur_row * img_row + cur_col];
					gradient_y = sobel_y_vec[cur_row * img_row + cur_col];
				}
				gradient_x_sum += gradient_x;
				gradient_y_sum += gradient_y;
				gradient_xx_sum += gradient_x * gradient_x;
				gradient_yy_sum += gradient_y * gradient_y;
				gradient_xy_sum += gradient_x * gradient_y;
			}
		}
		float size = (float) block_size * block_size;
		float cov_mat_00 = gradient_xx_sum / size  -
			gradient_x_sum * gradient_x_sum / size / size;
		float cov_mat_01 = gradient_xy_sum / size -
			gradient_x_sum * gradient_y_sum / size / size;
		float cov_mat_11 = gradient_yy_sum / size;
		// determinent - k * trace() ^ 2
		float discri_cond = cov_mat_00 * cov_mat_11 - cov_mat_01 * cov_mat_01 - 
			prop * (cov_mat_00 + cov_mat_11) * (cov_mat_00 + cov_mat_11);
		result_vec[index] = discri_cond;
		if (index == 0) 
		{
			printf("result_vec[0] = %f", result_vec[index]);
		}
	}
}


#include "hip/hip_runtime.h"
#pragma once
#include "../include/harris.h"
#include "opencv2/opencv.hpp"
#include "../include/conv.h"
#include "../include/utils.h"
#include <math.h>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__));


// cornerHarris��������ÿһ�����أ�x,y)��blockSize x blockSize �����ڣ�
// ����2x2�ݶȵ�Э�������M(x,y)���Ϳ����ҳ����ͼ�еľֲ����ֵ�����ҳ��˽ǵ㡣

namespace harris 
{
	void opencv_harris(const char * address)
	{
		cv::Mat src_img = cv::imread(address);
		cv::cvtColor(src_img, src_img, cv::COLOR_BGR2GRAY);
		cv::Mat harris_img;
		cornerHarris(src_img, harris_img, 2, 3, 0.04, cv::BORDER_DEFAULT);
		// harris_img type = CV_32F
		printf("type = %d", harris_img.type());
		cv::Mat harris_bw_img;
		cv::threshold(harris_img, harris_bw_img, 0.00001, 255, cv::THRESH_BINARY);
		cv::namedWindow("bw", cv::WINDOW_NORMAL);
		cv::imshow("bw", harris_bw_img);

		cv::normalize(harris_img, harris_img, 0, 1, cv::NORM_MINMAX, CV_32FC1);

		cv::namedWindow("harris_img", cv::WINDOW_NORMAL);
		cv::imshow("harris_img", harris_img);
		cv::waitKey(0);
	}

	void cuda_harris(cv::Mat & src, cv::Mat & dst, const int & block_size, const float & prop, 
		const int &aperture_size)
	{
		
		hipSetDevice(0);
		// 3 x 3 Sobel operator
		//float * sobel_y_kernel = new float[aperture_size * aperture_size];
		float * sobel_y = (float *)malloc(aperture_size * aperture_size * sizeof(float));
		sobel_y[0] = -1.; sobel_y[1] = -2.; sobel_y[2] = -1.;
		sobel_y[3] =  0.; sobel_y[4] = 0.; sobel_y[5] = 0.;
		sobel_y[6] =  1.; sobel_y[7] = 2.; sobel_y[8] = 1.;
		//float * sobel_x_kernel = new float[aperture_size * aperture_size];
		float * sobel_x = (float *)malloc(aperture_size * aperture_size * sizeof(float));
		sobel_x[0] = -1.; sobel_x[1] = 0.; sobel_x[2] = 1.;
		sobel_x[3] = -2.; sobel_x[4] = 0.; sobel_x[5] = 2.;
		sobel_x[6] = -1.; sobel_x[7] = 0.; sobel_x[8] = 1.;
		
		// conv get gradient map.
		cv::Mat sobel_x_img;
		conv::cuda_conv(src, sobel_x_img, sobel_x, aperture_size);
		cv::Mat sobel_y_img;
		conv::cuda_conv(src, sobel_y_img, sobel_y, aperture_size);
		printf("conv get gradient map finish \n");
		printf("sobel_x_img.type = %d", sobel_x_img.type());

		// /////////////////////////////////////////////////////////////////////////////////////
		cv::namedWindow("sobel_x", cv::WINDOW_NORMAL);
		cv::imshow("sobel_x", sobel_x_img);
		cv::waitKey(0);



		int img_rows = src.rows;
		int img_cols = src.cols;
		printf("\n src_img size = [%d, %d]\n", img_rows, img_cols);
		printf("\n gradient_img size = [%d, %d]\n", sobel_x_img.rows, sobel_x_img.cols);


		size_t img_size_float = img_cols * img_rows * sizeof(float);
		float * sobel_x_img_vec = NULL;
		float * sobel_y_img_vec = NULL;
		float * gpu_result_vec = NULL;

		// memory allocate
		HANDLE_ERROR(hipMalloc((void **)& sobel_x_img_vec, img_size_float));
		HANDLE_ERROR(hipMalloc((void **)& sobel_y_img_vec, img_size_float));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result_vec, img_size_float));

		// memory copy 
		HANDLE_ERROR(hipMemcpy(sobel_x_img_vec, (float *)sobel_x_img.data, img_size_float, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(sobel_y_img_vec, (float *)sobel_y_img.data, img_size_float, hipMemcpyHostToDevice));


		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		printf("block_num = %d, thread_num = %d !\n", block_num, thread_num);
		//dim3 thread_grid_size(block_num, 1, 1);
		//dim3 thread_block_size(thread_num, 1, 1);
		
		harris::harris_kernel<<< block_num, thread_num >>>
			(sobel_x_img_vec, sobel_y_img_vec, gpu_result_vec, img_rows, img_cols, block_size, prop);
		
		
		float * cpu_result_vec = (float *)malloc(img_size_float);
		HANDLE_ERROR(hipMemcpy(cpu_result_vec, gpu_result_vec, img_size_float, hipMemcpyDeviceToHost));

		dst = cv::Mat(img_rows, img_cols, CV_32FC1, cpu_result_vec).clone();
		
		HANDLE_ERROR(hipFree(sobel_x_img_vec));
		HANDLE_ERROR(hipFree(sobel_y_img_vec));
		HANDLE_ERROR(hipFree(gpu_result_vec));
		free(cpu_result_vec);
		free(sobel_y);
		free(sobel_x);
		hipDeviceReset();
	}

	void cuda_harris(float * src, float * dst, const int & img_rows, const int & img_cols, const int & block_size,
		const float & prop, const int &aperture_size) 
	{
		hipSetDevice(0);
		// 3 x 3 Sobel operator
		//float * sobel_y_kernel = new float[aperture_size * aperture_size];
		float * sobel_y_kernel = (float *)malloc(aperture_size * aperture_size * sizeof(float));
		sobel_y_kernel[0] = -1.; sobel_y_kernel[1] = -2.; sobel_y_kernel[2] = -1.;
		sobel_y_kernel[3] = 0.; sobel_y_kernel[4] = 0.; sobel_y_kernel[5] = 0.;
		sobel_y_kernel[6] = 1.; sobel_y_kernel[7] = 2.; sobel_y_kernel[8] = 1.;
		//float * sobel_x_kernel = new float[aperture_size * aperture_size];
		float * sobel_x_kernel = (float *)malloc(aperture_size * aperture_size * sizeof(float));
		sobel_x_kernel[0] = -1.; sobel_x_kernel[1] = 0.; sobel_x_kernel[2] = 1.;
		sobel_x_kernel[3] = -2.; sobel_x_kernel[4] = 0.; sobel_x_kernel[5] = 2.;
		sobel_x_kernel[6] = -1.; sobel_x_kernel[7] = 0.; sobel_x_kernel[8] = 1.;


		// conv get gradient map.
		size_t img_size_float = img_cols * img_rows * sizeof(float);

		float * sobel_x_img = (float *)malloc(img_size_float);

		conv::cuda_conv(src, sobel_x_img, img_rows, img_cols, sobel_x_kernel, aperture_size);

		float * sobel_y_img = (float *)malloc(img_size_float);
		conv::cuda_conv(src, sobel_y_img, img_rows, img_cols, sobel_y_kernel, aperture_size);

	
		float * sobel_x_img_vec = NULL;
		float * sobel_y_img_vec = NULL;
		float * gpu_result_vec = NULL;

		// memory allocate
		HANDLE_ERROR(hipMalloc((void **)& sobel_x_img_vec, img_size_float));
		HANDLE_ERROR(hipMalloc((void **)& sobel_y_img_vec, img_size_float));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result_vec, img_size_float));

		// memory copy 
		HANDLE_ERROR(hipMemcpy(sobel_x_img_vec, sobel_x_img, img_size_float, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(sobel_y_img_vec, sobel_y_img, img_size_float, hipMemcpyHostToDevice));


		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		printf("block_num = %d, thread_num = %d !\n", block_num, thread_num);
		//dim3 thread_grid_size(block_num, 1, 1);
		//dim3 thread_block_size(thread_num, 1, 1);

		harris::harris_kernel << < block_num, thread_num >> >
			(sobel_x_img_vec, sobel_y_img_vec, gpu_result_vec, img_rows, img_cols, block_size, prop);


		HANDLE_ERROR(hipMemcpy(dst, gpu_result_vec, img_size_float, hipMemcpyDeviceToHost));

		// //////////////////////////////////////////////////////////////////////
		// release memory
		// /////////////////////////////////////////////////////////////////////
		HANDLE_ERROR(hipFree(sobel_x_img_vec));
		HANDLE_ERROR(hipFree(sobel_y_img_vec));
		HANDLE_ERROR(hipFree(gpu_result_vec));

		free(sobel_y_kernel);
		free(sobel_x_kernel);
		free(sobel_x_img);
		free(sobel_y_img);
		
		hipDeviceReset();
	};


	__global__ void harris_kernel(float * sobel_x_vec, float * sobel_y_vec, float * gpu_result_vec, 
		const int img_row, const int img_col, const int block_size, 
		const float prop) 
	{
		
		int thread_id = threadIdx.x;
		int block_id = blockIdx.x;
		int index = block_id * blockDim.x + thread_id;
		
		int pixel_col = index % img_col;
		int pixel_row = index / img_col;
		// calculate covariance matrix
		float gradient_x_mean = 0;
		float gradient_y_mean = 0;
		float gradient_xy_mean = 0;
		float gradient_xx_mean = 0;
		float gradient_yy_mean = 0;
		float num = block_size * block_size;

		for (int i = 0; i < block_size; ++i)
		{
			for (int j = 0; j < block_size; ++j) 
			{
				int cur_row = pixel_row - block_size / 2 + i;
				int cur_col = pixel_col - block_size / 2 + j;
				//float 
				float gradient_x = 0.;
				float gradient_y = 0.;
				if (cur_row < 0 || cur_row >= img_row || cur_col < 0 || cur_col >= img_col) 
				{
				}
				else {
					gradient_x = sobel_x_vec[cur_row * img_col + cur_col];
					gradient_y = sobel_y_vec[cur_row * img_col + cur_col];
				}
				gradient_x_mean += gradient_x ;
				gradient_y_mean += gradient_y ;
				gradient_xx_mean += gradient_x * gradient_x ;
				gradient_yy_mean += gradient_y * gradient_y ;
				gradient_xy_mean += gradient_x * gradient_y ;
			}
		}
		float cov_mat_00 = gradient_xx_mean;
		float cov_mat_01 = gradient_xy_mean;
		float cov_mat_11 = gradient_yy_mean;
		// det(H) - k * trace(H) ^ 2, H represpent harris matrix
		float discri_cond = cov_mat_00 * cov_mat_11 - cov_mat_01 * cov_mat_01 - 
			prop * (cov_mat_00 + cov_mat_11) * (cov_mat_00 + cov_mat_11);
		gpu_result_vec[index] = discri_cond;
		if (index == 0) 
		{
			printf("result_vec[0] = %f", gpu_result_vec[index]);
		}
	}
}


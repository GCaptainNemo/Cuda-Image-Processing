#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/transform.h"
#include "../include/utils.h"
#include "pybind11/pybind11.h"

#include <opencv2/opencv.hpp>
#include <math.h>

// pytho API


#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__));

namespace transform {
	__global__ void transform_kernel(int * gpu_img, int * gpu_result, float * gpu_dst_grid_pos,
	float * gpu_homography_dst_to_src, const int img_cols, const int img_rows, const int grid_cols, const int grid_rows)
	{
		// rgb img as input(row dst_x col dst_x 3)
		// grid_pos((grid_row + 1) dst_x (grid_col + 1) dst_x 2)
		// homography(grid_row dst_x grid_col dst_x 9)
		int thread_id = threadIdx.x;
		int block_id = blockIdx.x;
		

		int pixel_id = block_id * blockDim.x + thread_id;
		if (pixel_id >= img_rows * img_cols)
		{
			return;
		}
		
		float dst_y = (float)(pixel_id / img_cols);  // row
		float dst_x = (float)(pixel_id% img_cols);  // col
		if (pixel_id == 12743)
		{	
			printf("pixel_id = %d	, dst_y = %f,	dst_x =%f", pixel_id, dst_y, dst_x);
			printf("img_cols = %d, img_rows = %d, grid_cols = %d, grid_rows = %d", img_cols, img_rows, grid_cols, grid_rows);
		}	
		for (int i = 0; i < 3; ++i)
		{
			gpu_result[pixel_id * 3 + i] = 0;
		}
		
		for (int i = 0; i < grid_rows; ++i)
		{
			for (int j = 0; j < grid_cols; ++j)
			{
				// grid points id(顶点并网格数多1)
				int id = i * (grid_cols + 1) + j;
				float left_up_delta_x = gpu_dst_grid_pos[id * 2] - dst_x;
				float left_up_delta_y = gpu_dst_grid_pos[id * 2 + 1] - dst_y;
				float left_down_delta_x = gpu_dst_grid_pos[(id + grid_cols + 1) * 2] - dst_x;
				float left_down_delta_y = gpu_dst_grid_pos[(id + grid_cols + 1) * 2 + 1] - dst_y;
				float right_down_delta_x = gpu_dst_grid_pos[(id + grid_cols + 2) * 2] - dst_x;
				float right_down_delta_y = gpu_dst_grid_pos[(id + grid_cols + 2) * 2 + 1] - dst_y;
				float right_up_delta_x = gpu_dst_grid_pos[(id + 1) * 2] - dst_x;
				float right_up_delta_y = gpu_dst_grid_pos[(id + 1) * 2 + 1] - dst_y;
				// ///////////////////////////////////////////////////////////////////////
				// cross product
				// ////////////////////////////////////////////////////////////////////////
				float prod_1 = left_up_delta_x * left_down_delta_y - left_up_delta_y * left_down_delta_x;
				if (prod_1 > 0) {
					continue;
				}
				float prod_2 = left_down_delta_x * right_down_delta_y - left_down_delta_y * right_down_delta_x;
				if (prod_2 >= 0) {
					continue;
				}
				float prod_3 = right_down_delta_x * right_up_delta_y - right_down_delta_y * right_up_delta_x;
				if (prod_3 >= 0) {
					continue;
				}
				float prod_4 = right_up_delta_x * left_up_delta_y - right_up_delta_y * left_up_delta_x;
				if (prod_4 > 0) {
					continue;
				}
				// //////////////////////////////////////////////////////////////////////////////
				// dst_x, dst_y inside grid
				// //////////////////////////////////////////////////////////////////////////////
				int homography_id = i * grid_cols + j;
				float src_x = gpu_homography_dst_to_src[homography_id * 9 + 0] * dst_x + 
							  gpu_homography_dst_to_src[homography_id * 9 + 1] * dst_y + 
							  gpu_homography_dst_to_src[homography_id * 9 + 2];
				float src_y = gpu_homography_dst_to_src[homography_id * 9 + 3] * dst_x + 
							  gpu_homography_dst_to_src[homography_id * 9 + 4] * dst_y + 
							  gpu_homography_dst_to_src[homography_id * 9 + 5];
				float nor_f = gpu_homography_dst_to_src[homography_id * 9 + 6] * dst_x + 
							  gpu_homography_dst_to_src[homography_id * 9 + 7] * dst_y + 
							  gpu_homography_dst_to_src[homography_id * 9 + 8];
				src_x = src_x / nor_f;
				src_y = src_y / nor_f;
				
				if (src_x >= 0 && src_x < img_cols && src_y >= 0 && src_y < img_rows)
				{
					// printf("src_x = %f, src_y = %f", src_x, src_y);
					int left_up_x = (int)src_x;
					int left_up_y = (int)src_y;
					// printf("left_up_x = %d, left_up_y = %d", left_up_x, left_up_y);
									
					int left_up_idx = (left_up_x + left_up_y * img_cols) * 3;
					int left_down_idx = (left_up_x + (left_up_y + 1)* img_cols) * 3;
					int right_down_idx = (left_up_x + 1 + (left_up_y + 1)* img_cols) * 3;
					int right_up_idx = (left_up_x + 1 + left_up_y * img_cols) * 3;

					float proportion_x = src_x - (float)left_up_x;
					float proportion_y = src_y - (float)left_up_y;
					for (int offset = 0; offset < 3; ++offset)
					{

						float res = (1 - proportion_x) * (1 - proportion_y) * gpu_img[left_up_idx + offset] +
							(1 - proportion_x) * proportion_y * gpu_img[left_down_idx + offset] +
							proportion_x * (1 - proportion_y) * gpu_img[right_up_idx + offset] +
							proportion_x * proportion_y * gpu_img[right_down_idx + offset];
						if (res > 255) {
							gpu_result[pixel_id * 3 + offset] = 255;
						}
						else {
							gpu_result[pixel_id * 3 + offset] = (int)res;
						}
					}
				}
				return;

			}
		}
	}
	
	

	void cuda_transform(cv::Mat & src, cv::Mat & dst, float * cpu_dst_grid_pos, float * cpu_src_grid_pos,
		float * cpu_homography_dst_to_src, int grid_cols, int grid_rows)
	{
		// read linshi and convert to gray image
		hipSetDevice(0);

		cv::Mat linshi;
		if (src.type() != CV_8UC3) {
			return;
		}
		else {
			linshi = src.clone();
		}
		int img_cols = linshi.cols;
		int img_rows = linshi.rows;


		int * gpu_img = NULL;
		int * gpu_result = NULL;
		float * gpu_dst_grid_pos = NULL;
		float * gpu_src_grid_pos = NULL;
		float * gpu_homography_dst_to_src = NULL;

		size_t img_size = img_cols * img_rows * sizeof(uint3);
		size_t grid_pos_size = (grid_cols + 1)* (grid_rows + 1) * 2 * sizeof(float);
		size_t homography_size = grid_cols * grid_rows * 9 * sizeof(float);

		HANDLE_ERROR(hipMalloc((void **)& gpu_img, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_dst_grid_pos, grid_pos_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_src_grid_pos, grid_pos_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_homography_dst_to_src, homography_size));

		// memory copy kernel and linshi from host to device
		HANDLE_ERROR(hipMemcpy(gpu_img, linshi.data, img_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_dst_grid_pos, cpu_dst_grid_pos, grid_pos_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_src_grid_pos, cpu_src_grid_pos, grid_pos_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_homography_dst_to_src, cpu_homography_dst_to_src, homography_size, hipMemcpyHostToDevice));

		// //////////////////////////////////////////////////////////////////////////////////////////////
		// resident thread; every pixel of result correspond to a thread;
		// //////////////////////////////////////////////////////////////////////////////////////////////

		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		dim3 grid_size(block_num, 1, 1);
		dim3 block_size(thread_num, 1, 1);
		transform::transform_kernel << < grid_size, block_size >> > (gpu_img, gpu_dst_grid_pos, gpu_src_grid_pos,
			gpu_homography_dst_to_src, gpu_result,
			img_cols, img_rows, grid_cols, grid_rows);

		float * cpu_result = new float[img_cols * img_rows * 3];
		HANDLE_ERROR(hipMemcpy(cpu_result, gpu_result, img_size, hipMemcpyDeviceToHost));

		dst = cv::Mat(img_rows, img_cols, CV_8UC3, cpu_result).clone();
		printf("row = 0, col=0, val = %f", dst.at<int>(0, 0));

		cv::normalize(dst, dst, 1.0, 0.0, cv::NORM_MINMAX);

		HANDLE_ERROR(hipFree(gpu_img));
		HANDLE_ERROR(hipFree(gpu_src_grid_pos));
		HANDLE_ERROR(hipFree(gpu_dst_grid_pos));
		HANDLE_ERROR(hipFree(gpu_homography_dst_to_src));
		HANDLE_ERROR(hipFree(gpu_result));
		delete[] cpu_result;
		hipDeviceReset();
	}

	void cuda_transform(int * src, int * dst, float * cpu_dst_grid_pos, float * cpu_src_grid_pos,
		float * cpu_homography_dst_to_src, int grid_cols, int grid_rows, int img_cols, int img_rows)
	{
		// read linshi and convert to gray image
		// read linshi and convert to gray image
		hipSetDevice(0);

		int * gpu_img = NULL;
		int * gpu_result = NULL;
		float * gpu_dst_grid_pos = NULL;
		float * gpu_src_grid_pos = NULL;
		float * gpu_homography_dst_to_src = NULL;

		size_t img_size = img_cols * img_rows * sizeof(uint3);
		size_t grid_pos_size = (grid_cols + 1)* (grid_rows + 1) * 2 * sizeof(float);
		size_t homography_size = grid_cols * grid_rows * 9 * sizeof(float);

		HANDLE_ERROR(hipMalloc((void **)& gpu_img, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_dst_grid_pos, grid_pos_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_src_grid_pos, grid_pos_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_homography_dst_to_src, homography_size));

		// memory copy kernel and linshi from host to device
		HANDLE_ERROR(hipMemcpy(gpu_img, src, img_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_dst_grid_pos, cpu_dst_grid_pos, grid_pos_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_src_grid_pos, cpu_src_grid_pos, grid_pos_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_homography_dst_to_src, cpu_homography_dst_to_src, homography_size, hipMemcpyHostToDevice));

		// //////////////////////////////////////////////////////////////////////////////////////////////
		// resident thread; every pixel of result correspond to a thread;
		// //////////////////////////////////////////////////////////////////////////////////////////////

		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		dim3 grid_size(block_num, 1, 1);
		dim3 block_size(thread_num, 1, 1);
		transform::transform_kernel << < grid_size, block_size >> > (gpu_img, gpu_dst_grid_pos, gpu_src_grid_pos,
			gpu_homography_dst_to_src, gpu_result,
			img_cols, img_rows, grid_cols, grid_rows);


		HANDLE_ERROR(hipMemcpy(dst, gpu_result, img_size, hipMemcpyDeviceToHost));

		HANDLE_ERROR(hipFree(gpu_img));
		HANDLE_ERROR(hipFree(gpu_src_grid_pos));
		HANDLE_ERROR(hipFree(gpu_dst_grid_pos));
		HANDLE_ERROR(hipFree(gpu_homography_dst_to_src));
		HANDLE_ERROR(hipFree(gpu_result));
		hipDeviceReset();
	}


}
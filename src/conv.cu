#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../include/conv.h"
#include "opencv2/opencv.hpp"
#include "../include/utils.h"
#include <math.h>

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__));

namespace conv {
	__global__ void conv_kernel(float *gpu_img, float * gpu_kernel, float * gpu_result,
		const int img_cols, const int img_rows, const int kernel_dim)
	{
		// the kernel anchor is at the center.
		int thread_id = threadIdx.x;
		int block_id = blockIdx.x;

		int pixel_id = block_id * blockDim.x + thread_id;
		if (pixel_id >= img_rows * img_cols)
		{
			return;
		}

		int row = pixel_id / img_cols;
		int col = pixel_id % img_cols;
		gpu_result[pixel_id] = 0;
		for (int i = 0; i < kernel_dim; ++i)
		{
			for (int j = 0; j < kernel_dim; ++j)
			{
				float img_val = 0;
				int cur_rows = row - kernel_dim / 2 + i;
				int cur_cols = col - kernel_dim / 2 + j;
				if (cur_cols < 0 || cur_rows < 0 || cur_cols >= img_cols || cur_rows >= img_rows)
				{
				}
				else
				{
					img_val = gpu_img[cur_cols + cur_rows * img_cols];
				}
				gpu_result[pixel_id] += (gpu_kernel[i * kernel_dim + j]) * img_val;
			}
		}
	}

	void cuda_conv(cv::Mat & src, cv::Mat & dst, float * kernel, int kernel_dim)
	{
		// read linshi and convert to gray image
		hipSetDevice(0);
		
		cv::Mat linshi;
 		if (src.type() == CV_8UC3) {
			cv::cvtColor(src, linshi, cv::COLOR_BGR2GRAY);
			printf("origin gray img\n");
			// uchar to float
			linshi.convertTo(linshi, CV_32FC1);
		}
		else {
			linshi = src.clone();
		}
		int img_cols = linshi.cols;
		int img_rows = linshi.rows;


		float * gpu_img = NULL;
		float * gpu_result = NULL;
		float * gpu_kernel = NULL;


		size_t img_size = img_cols * img_rows * sizeof(float);
		size_t kernel_size = kernel_dim * kernel_dim * sizeof(float);

		HANDLE_ERROR(hipMalloc((void **)& gpu_img, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result, img_size));
		HANDLE_ERROR(hipMalloc((void **)& gpu_kernel, kernel_size));
		// memory copy kernel and linshi from host to device
		HANDLE_ERROR(hipMemcpy(gpu_img, linshi.data, img_size, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_kernel, kernel, kernel_size, hipMemcpyHostToDevice));

		// //////////////////////////////////////////////////////////////////////////////////////////////
		// resident thread; every pixel of result correspond to a thread;
		// //////////////////////////////////////////////////////////////////////////////////////////////

		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		dim3 grid_size(block_num, 1, 1);
		dim3 block_size(thread_num, 1, 1);
		conv::conv_kernel <<< grid_size, block_size >> > (gpu_img, gpu_kernel, gpu_result, img_cols, img_rows, kernel_dim);
		
		float * cpu_result = new float[img_cols * img_rows];
		HANDLE_ERROR(hipMemcpy(cpu_result, gpu_result, img_size, hipMemcpyDeviceToHost));
		
		dst = cv::Mat(img_rows, img_cols, CV_32FC1, cpu_result).clone();
		printf("row = 0, col=0, val = %f", dst.at<float>(0, 0));

		cv::normalize(dst, dst, 1.0, 0.0, cv::NORM_MINMAX);

		HANDLE_ERROR(hipFree(gpu_img));
		HANDLE_ERROR(hipFree(gpu_kernel));
		HANDLE_ERROR(hipFree(gpu_result));
		delete [] cpu_result;
		hipDeviceReset();
	}

	void cuda_conv(float * src, float * dst, int img_rows, int img_cols, float * kernel, int kernel_dim) 
	{
		// read linshi and convert to gray image
		hipSetDevice(0);

		float * gpu_img = NULL;
		float * gpu_result = NULL;
		float * gpu_kernel = NULL;


		size_t img_size_t = img_cols * img_rows * sizeof(float);
		size_t kernel_size_t = kernel_dim * kernel_dim * sizeof(float);

		HANDLE_ERROR(hipMalloc((void **)& gpu_img, img_size_t));
		HANDLE_ERROR(hipMalloc((void **)& gpu_result, img_size_t));
		HANDLE_ERROR(hipMalloc((void **)& gpu_kernel, kernel_size_t));
		// memory copy kernel and linshi from host to device
		HANDLE_ERROR(hipMemcpy(gpu_img, (float *)src, img_size_t, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(gpu_kernel, kernel, kernel_size_t, hipMemcpyHostToDevice));

		// //////////////////////////////////////////////////////////////////////////////////////////////
		// resident thread; every pixel of result correspond to a thread;
		// //////////////////////////////////////////////////////////////////////////////////////////////

		int thread_num = getThreadNum();
		int block_num = (img_cols * img_rows - 0.5) / thread_num + 1;
		dim3 grid_size(block_num, 1, 1);
		dim3 block_size(thread_num, 1, 1);
		conv::conv_kernel << < grid_size, block_size >> > (gpu_img, gpu_kernel, gpu_result, img_cols, img_rows, kernel_dim);

		HANDLE_ERROR(hipMemcpy(dst, gpu_result, img_size_t, hipMemcpyDeviceToHost));

		// release gpu memory
		HANDLE_ERROR(hipFree(gpu_img));
		HANDLE_ERROR(hipFree(gpu_kernel));
		HANDLE_ERROR(hipFree(gpu_result));
		hipDeviceReset();
	};


	void opencv_conv(const char * address)
	{
		cv::Mat kernel_ = (cv::Mat_<float>(3, 3) << -1, 0, 1, -1, 0, 1, -1, 0, 1);
		cv::Mat src_img = cv::imread(address);
		cv::cvtColor(src_img, src_img, cv::COLOR_BGR2GRAY);
		src_img.convertTo(src_img, CV_32FC1);
		cv::Mat dst_img;
		cv::filter2D(src_img, dst_img, src_img.depth(), kernel_);
		cv::normalize(dst_img, dst_img, 1.0, 0.0, cv::NORM_MINMAX);
		cv::namedWindow("dst_img", cv::WINDOW_NORMAL);
		cv::imshow("dst_img", dst_img);
		cv::waitKey(0);
	}
}